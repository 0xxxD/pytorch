
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathReduce.cu"
#else

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t value, int dimension, scalar_t maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  dimension = at::maybe_wrap_dim(dimension, src);
  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<scalar_t>::gt(value, scalar_cast<scalar_t>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    // NOTE: only with this specific number of threads can this work on GPUs with a warp size != 32 (such as AMD). Do not alter w/o changing buffer size in kernel.
    dim3 threads(32);

    THCTensor_kernel_renorm<scalar_t, accreal>
      <<<grid, threads, 0, c10::cuda::getCurrentCUDAStream()>>>
      (THCTensor_(data)(state, data), scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    cudaError_t errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

accreal THCTensor_(std_all)(THCState *state, THCTensor *self, bool unbiased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCNumerics<accreal>::sqrt((THCTensor_(var_all)(state, self, unbiased)));
}

accreal THCTensor_(var_all)(THCState *state, THCTensor *self, bool unbiased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal mean = THTensor_wrap(self).mean().item<accreal>();

  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           SquareFunctor<accreal>(mean),
                           ReduceAdd<accreal>(),
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  val = THCNumerics<accreal>::div(
    val,
    scalar_cast<accreal>(std::max<int64_t>(0, THCTensor_(nElement)(state, self) - (unbiased ? 1 : 0)))
  );

  THCudaCheck(cudaGetLastError());
  return val;
}

#endif

#endif

#endif
